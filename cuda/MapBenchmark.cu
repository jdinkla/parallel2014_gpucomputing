#include "hip/hip_runtime.h"
/*
* Copyright (c) 2014 by Joern Dinkla, www.dinkla.com, All rights reserved.
*
* See the LICENSE file in the root directory.
*/

#include <thrust/device_vector.h>
#include <thrust/sequence.h>
#include "MapBenchmark.h"
#include "CudaTimer.h"
#include "Extent.h"
#include "CudaUtilities.h"
#include "CudaExecConfig.h"
#include "IdFunctor.h"
#include "Map.h"
#include <iostream>
#include "IdFunctor.h"

using namespace std;

// Diese Gr��en �ndern, wenn der Speicherplatz nicht ausreicht.
const int sizeX = 14 * 1024;
const int sizeY = 14 * 1024;

template <typename T>
void bench_map_copy_timed(
	CudaTimer& timer,
	Extent& ext,
	thrust::device_vector<T>& d1,
	thrust::device_vector<T>& d2,
	dim3 thr)
{
	timer.start();

	// Rufe Kernel auf und sync
	CudaExecConfig cnf(ext, thr);
	IdFunctor<T> op;
	map(cnf, op, ext, d1, d2);
	hipDeviceSynchronize();
	check_cuda();

	timer.stop();

	cout
		<< thr.x
		<< ";" << ext.get_number_of_elems() * sizeof(T)
		<< ";" << timer.delta()
		<< endl;
}

template <typename T>
void bench_map(Extent& ext, thrust::device_vector<T>& d1, thrust::device_vector<T>& d2)
{
	CudaTimer timer;
	hipDeviceSynchronize();
	check_cuda();
	for (int t = 1; t <= 1024; t++)
	{
		bench_map_copy_timed<T>(timer, ext, d1, d2, dim3(t, 1, 1));
	}
}

template <typename T>
void bench_map_32(Extent& ext, thrust::device_vector<T>& d1, thrust::device_vector<T>& d2)
{
	CudaTimer timer;
	hipDeviceSynchronize();
	check_cuda();
	for (int t = 32; t <= 1024; t += 32)
	{
		bench_map_copy_timed<T>(timer, ext, d1, d2, dim3(t, 1, 1));
	}
}

template <typename T>
void bench_map_32_short(Extent& ext, thrust::device_vector<T>& d1, thrust::device_vector<T>& d2)
{
	CudaTimer timer;
	hipDeviceSynchronize();
	check_cuda();
	for (int t = 32; t <= 256; t += 32)
	{
		bench_map_copy_timed<T>(timer, ext, d1, d2, dim3(t, 1, 1));
	}
}

void bench_map_copy()
{
	CudaTimer timer;
	Extent ext(sizeX* sizeY);
	thrust::device_vector<int> d1(ext.get_number_of_elems());
	thrust::sequence(d1.begin(), d1.end(), 0);
	thrust::device_vector<int> d2(ext.get_number_of_elems());
	hipDeviceSynchronize();
	check_cuda();
	bench_map<int>(ext, d1, d2);
}

void bench_map_copy_32()
{
	CudaTimer timer;
	Extent ext(sizeX* sizeY);
	thrust::device_vector<int> d1(ext.get_number_of_elems());
	thrust::sequence(d1.begin(), d1.end(), 0);
	thrust::device_vector<int> d2(ext.get_number_of_elems());
	hipDeviceSynchronize();
	check_cuda();
	bench_map_32<int>(ext, d1, d2);
}

void bench_map_copy_32_short()
{
	CudaTimer timer;
	Extent ext(sizeX* sizeY);
	thrust::device_vector<int> d1(ext.get_number_of_elems());
	thrust::sequence(d1.begin(), d1.end(), 0);
	thrust::device_vector<int> d2(ext.get_number_of_elems());
	hipDeviceSynchronize();
	check_cuda();
	bench_map_32_short<int>(ext, d1, d2);
}

void bench_map_copy_int2()
{
	CudaTimer timer;
	Extent ext(sizeX / 2 * sizeY);
	thrust::device_vector<int2> d1(ext.get_number_of_elems());
	thrust::device_vector<int2> d2(ext.get_number_of_elems());
	hipDeviceSynchronize();
	check_cuda();
	bench_map<int2>(ext, d1, d2);
}

void bench_map_copy_int2_32()
{
	CudaTimer timer;
	Extent ext(sizeX / 2 * sizeY);
	thrust::device_vector<int2> d1(ext.get_number_of_elems());
	thrust::device_vector<int2> d2(ext.get_number_of_elems());
	hipDeviceSynchronize();
	check_cuda();
	bench_map_32<int2>(ext, d1, d2);
}

void bench_map_copy_int2_32_short()
{
	CudaTimer timer;
	Extent ext(sizeX / 2 * sizeY);
	thrust::device_vector<int2> d1(ext.get_number_of_elems());
	thrust::device_vector<int2> d2(ext.get_number_of_elems());
	hipDeviceSynchronize();
	check_cuda();
	bench_map_32_short<int2>(ext, d1, d2);
}

void bench_map_copy_int4()
{
	CudaTimer timer;
	Extent ext(sizeX / 4 * sizeY);
	thrust::device_vector<int4> d1(ext.get_number_of_elems());
	thrust::device_vector<int4> d2(ext.get_number_of_elems());
	hipDeviceSynchronize();
	check_cuda();
	bench_map<int4>(ext, d1, d2);
}

void bench_map_copy_int4_32()
{
	CudaTimer timer;
	Extent ext(sizeX / 4 * sizeY);
	thrust::device_vector<int4> d1(ext.get_number_of_elems());
	thrust::device_vector<int4> d2(ext.get_number_of_elems());
	hipDeviceSynchronize();
	check_cuda();
	bench_map_32<int4>(ext, d1, d2);
}

void bench_map_copy_int4_32_short()
{
	CudaTimer timer;
	Extent ext(sizeX / 4 * sizeY);
	thrust::device_vector<int4> d1(ext.get_number_of_elems());
	thrust::device_vector<int4> d2(ext.get_number_of_elems());
	hipDeviceSynchronize();
	check_cuda();
	bench_map_32_short<int4>(ext, d1, d2);
}
